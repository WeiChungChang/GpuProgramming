#include <stdio.h>
#include <hip/hip_runtime.h>

#define HANDLE_ERROR(apiFuncCall)                                              \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define HANDLE_NULL(x)
#define TOTAL 1024

float cuda_malloc_test( int size, bool up ) {
    hipEvent_t start, stop;
    int *a, *dev_a;
    float elapsedTime = 0.0f;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    a = (int*)malloc( size * sizeof( *a ) );
    if (!a) {
        exit(-1);
    }
    HANDLE_ERROR( hipMalloc( (void**)&dev_a,
    size * sizeof( *dev_a ) ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );
    for (int i=0; i<TOTAL; i++) {
    if (up)
        HANDLE_ERROR( hipMemcpy( dev_a, a, size * sizeof( *dev_a ),hipMemcpyHostToDevice ) );
    else
        HANDLE_ERROR( hipMemcpy( a, dev_a,size * sizeof( *dev_a ),hipMemcpyDeviceToHost ) );
    }
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,start, stop ) );
    free( a );
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );
    return elapsedTime;
}

float cuda_host_alloc_test( int size, bool up ) {
    hipEvent_t start, stop;int *a, *dev_a;
    float elapsedTime;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&a, size * sizeof( *a ),hipHostMallocDefault ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_a,size * sizeof( *dev_a ) ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );
    for (int i=0; i<TOTAL; i++) {
        if (up)
        HANDLE_ERROR( hipMemcpy( dev_a, a, size * sizeof( *a ),hipMemcpyHostToDevice ) );
        else
        HANDLE_ERROR( hipMemcpy( a, dev_a,size * sizeof( *a ),hipMemcpyDeviceToHost ) );
    }
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,start, stop ) );
    HANDLE_ERROR( hipHostFree( a ) );
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );
    return elapsedTime;
}

//#define SIZE    (10*1024*1024)
//#define SIZE    (1*1024)
//#define SIZE    (4*1024)
//#define SIZE    (16*1024)
//#define SIZE      (64*1024)
//#define SIZE    (1024*1024)
#define SIZE    (4*1024*1024)
//#define SIZE    (16*1024*1024)


int main( void ) {
    float elapsedTime;
    float MB = (float)TOTAL*SIZE*sizeof(int)/1024/1024;

    printf( "Pinned:\n" );
    elapsedTime = cuda_host_alloc_test( SIZE, true );
    printf( "Up Time using cudaHostAlloc:%3.1f ms\n",elapsedTime );
    printf( "\tTransfer %d Bytes;  MB/s during copy down:%3.1f\n", SIZE*4, MB /(elapsedTime/1000) );

    elapsedTime = cuda_host_alloc_test( SIZE, false );
    printf( "Down Time using cudaHostAlloc:%3.1f ms\n",elapsedTime );
    printf( "\tTransfer %d Bytes;  MB/s during copy down:%3.1f\n", SIZE*4, MB /(elapsedTime/1000) );

    printf("\n-------------------------------\n\n");

    printf( "Native:\n" );
    elapsedTime = cuda_malloc_test( SIZE, true );
    printf( "Up Time using cudaMalloc:%3.1f ms\n",elapsedTime );
    printf( "\tTransfer %d Bytes;  MB/s during copy down:%3.1f\n", SIZE*4, MB /(elapsedTime/1000) );
    
    elapsedTime = cuda_malloc_test( SIZE, false );
    printf( "Down Time using cudaMalloc:%3.1f ms\n",elapsedTime );
    printf( "\tTransfer %d	 Bytes;  MB/s during copy down:%3.1f\n", SIZE*4, MB /(elapsedTime/1000) );
	
}